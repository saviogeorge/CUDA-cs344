#include "hip/hip_runtime.h"
// Homework 1
// Color to Greyscale Conversion

//A common way to represent color images is known as RGBA - the color
//is specified by how much Red, Green, and Blue is in it.
//The 'A' stands for Alpha and is used for transparency; it will be
//ignored in this homework.

//Each channel Red, Blue, Green, and Alpha is represented by one byte.
//Since we are using one byte for each color there are 256 different
//possible values for each color.  This means we use 4 bytes per pixel.

//Greyscale images are represented by a single intensity value per pixel
//which is one byte in size.

//To convert an image from color to grayscale one simple method is to
//set the intensity to the average of the RGB channels.  But we will
//use a more sophisticated method that takes into account how the eye 
//perceives color and weights the channels unequally.

//The eye responds most strongly to green followed by red and then blue.
//The NTSC (National Television System Committee) recommends the following
//formula for color to greyscale conversion:

//I = .299f * R + .587f * G + .114f * B

//Notice the trailing f's on the numbers which indicate that they are 
//single precision floating point constants and not double precision
//constants.

//You should fill in the kernel as well as set the block and grid sizes
//so that the entire image is processed.

//#include "reference_calc.cpp"
#include "utils.h"
#include <stdio.h>
#include ""

// Global functions are also called "kernels". 
// It's the functions that you may call from the 
// host side using CUDA kernel call semantics
__global__
void rgba_to_greyscale(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx_x < numRows && idx_y < numCols) {
        int idx_1d = idx_x * numCols + idx_y;
        const uchar4 rgba = rgbaImage[idx_1d];
        float fpix = .299f * rgba.x + .587f * rgba.y + .114f * rgba.z;
        greyImage[idx_1d] = fpix;
    }
}

void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
                            unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{

    const int thread = 16;
    const dim3 blockSize(thread, thread, 1);
    const dim3 gridSize(ceil(numRows /(float)thread), ceil(numCols /(float)thread), 1);
    rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
  
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}

// Notes:
// https://stackoverflow.com/questions/4391162/cuda-determining-threads-per-block-blocks-per-grid
// https://stackoverflow.com/questions/12373940/difference-between-global-and-device-functions